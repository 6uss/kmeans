#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 25


/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {


  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              idx, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states,  unsigned float* numbers) {
  /* hiprand works like rand - except that it takes a state as a parameter */
int idx = threadIdx.x + blockDim.x*blockIdx.x;
    float myrandf = hiprand_uniform(my_curandstate+idx);
       
    myrandf *= (max_rand_int[idx] - min_rand_int[idx]+0.999999);
    myrandf += min_rand_int[idx];
  numbers[blockIdx.x] = myrandf;
}

int main( ) {
  /* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t* states;

  unsigned *d_max;
  dma
  /* allocate space on the GPU for the random states */
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));

  /* invoke the GPU to initialize all of the random states */
  init<<<N, 1>>>(time(0), states);

  /* allocate an array of unsigned ints on the CPU and GPU */
  unsigned int cpu_nums[N];
  unsigned float* gpu_nums;
  hipMalloc((void**) &gpu_nums, N * sizeof(unsigned int));

  /* invoke the kernel to get some random numbers */
  randoms<<<N, 1>>>(states, gpu_nums);

  /* copy the random numbers back */
  hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

  /* print them out */
  for (int i = 0; i < N; i++) {
    printf("%u\n", cpu_nums[i]);
  }

  /* free the memory we allocated for the states and numbers */
  hipFree(states);
  hipFree(gpu_nums);

  return 0;
}