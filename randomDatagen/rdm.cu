#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
#define MIN 500
#define MAX 600

// #define ITER 10000000

#define DSIZE 72
#define nTPB 256


__global__ void setup_kernel(hiprandState *state,unsigned long seed){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);
}
// __global__ void generate_kernel(hiprandState *my_curandstate, const unsigned int n, 
// const unsigned *max_rand_int, const unsigned *min_rand_int,  unsigned int *result){

__global__ void randfloat(hiprandState *state,const unsigned *min_rand_int,
			const unsigned *max_rand_int, float *result)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
  	
		float myrandf = hiprand_uniform(state);
		myrandf *= (max_rand_int[idx] - min_rand_int[idx]+ 0.999999);
		myrandf += min_rand_int[idx];
		
	    result[idx]=myrandf;
	
}

int main(){

  hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));
  float *d_result, *h_result;
  unsigned *d_max_rand_int, *h_max_rand_int, *d_min_rand_int, *h_min_rand_int;

  hipMalloc(&d_result, (MAX-MIN+1)* sizeof(float));
  h_result = (float *)malloc(((MAX-MIN+1))*sizeof(float));

    
  hipMalloc(&d_max_rand_int, sizeof(unsigned));
  h_max_rand_int = (unsigned *)malloc(sizeof(unsigned));
  hipMalloc(&d_min_rand_int, sizeof(unsigned));
  h_min_rand_int = (unsigned *)malloc(sizeof(unsigned));

  hipMemset(d_result, 0, (MAX-MIN+1)*sizeof(float));
  setup_kernel<<<5,5>>>(d_state,time(NULL));
  *h_max_rand_int = MAX;
  *h_min_rand_int = MIN;

  hipMemcpy(d_max_rand_int, h_max_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(d_min_rand_int, h_min_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);

  randfloat<<<2,18>>>(d_state,d_min_rand_int,d_max_rand_int, d_result);

  hipMemcpy(h_result, d_result,(MAX-MIN+1) * sizeof(float), hipMemcpyDeviceToHost);
  int index=0;
  for (int i = 0; i < (MAX-MIN+1);index++)
  {
    printf("%d %.6f %.6f %.6f %.6f %.6f %.6f %.6f %.6f %.6f \n" ,index,
     h_result[i], h_result[i+1], h_result[i+2],
     h_result[i+3], h_result[i+4], h_result[i+5],
     h_result[i+6], h_result[i+7], h_result[i+9]);
    i=i+9;

  }



/* 
  hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));
  unsigned *d_result, *h_result;
  unsigned *d_max_rand_int, *h_max_rand_int, *d_min_rand_int, *h_min_rand_int;
  hipMalloc(&d_result, (MAX-MIN+1) * sizeof(unsigned));
  h_result = (unsigned *)malloc((MAX-MIN+1)*sizeof(unsigned));
  hipMalloc(&d_max_rand_int, sizeof(unsigned));
  h_max_rand_int = (unsigned *)malloc(sizeof(unsigned));
  hipMalloc(&d_min_rand_int, sizeof(unsigned));
  h_min_rand_int = (unsigned *)malloc(sizeof(unsigned));
  hipMemset(d_result, 0, (MAX-MIN+1)*sizeof(unsigned));
  setup_kernel<<<1,1>>>(d_state,time(NULL));

  *h_max_rand_int = MAX;
  *h_min_rand_int = MIN;
  hipMemcpy(d_max_rand_int, h_max_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(d_min_rand_int, h_min_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);
  generate_kernel<<<1,1>>>(d_state, ITER, d_max_rand_int, d_min_rand_int, d_result);
  hipMemcpy(h_result, d_result, (MAX-MIN+1) * sizeof(unsigned), hipMemcpyDeviceToHost);
  printf("Bin:    Count: \n");
  for (int i = MIN; i <= MAX; i++)
    printf("%d    %d\n", i, h_result[i-MIN]);

  return 0;*/
}


// $ nvcc -arch=sm_20 -o t527 t527.cu -lcurand
// $ cuda-memcheck ./t527
// ========= CUDA-MEMCHECK
// Bin:    Count:
// 2    1665496
// 3    1668130
// 4    1667644
// 5    1667435
// 6    1665026
// 7    1666269
// ========= ERROR SUMMARY: 0 errors
// $