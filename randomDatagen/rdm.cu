#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
#define MIN 2
#define MAX 7

// #define ITER 10000000

#define DSIZE 10000000
#define nTPB 256


__global__ void setup_kernel(hiprandState *state,unsigned long seed){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void randfloat(hiprandState *state, float *result){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    result[idx] = hiprand_normal(&state[idx]+idx);
}
/*
__global__ void generate_kernel(hiprandState *my_curandstate, const unsigned int n,
                const unsigned *max_rand_int, const unsigned *min_rand_int,  unsigned int *result){

  int idx = threadIdx.x + blockDim.x*blockIdx.x;

  int count = 0;
  while (count < n){
    float myrandf = hiprand_normal(my_curandstate+idx);
    myrandf *= (max_rand_int[idx] - min_rand_int[idx]+0.999999);
    myrandf += min_rand_int[idx];
    int myrand = (int)truncf(myrandf);

    assert(myrand <= max_rand_int[idx]);
    assert(myrand >= min_rand_int[idx]);
    result[myrand-min_rand_int[idx]]++;
    count++;}
}
*/
int main(){


  // hiprandState *devState; 
  // int n_iter = 20;
  // float *d_a;             

  // gpuErrchk(hipMalloc((void**)&devState, DSIZE*sizeof(hiprandState)));

  // gpuErrchk(hipMalloc((void**)&d_a, DSIZE*sizeof(float)));

  hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));
  float *d_result, *h_result;
  // unsigned *d_max_rand_int, *h_max_rand_int, *d_min_rand_int, *h_min_rand_int;

  hipMalloc(&d_result, DSIZE* sizeof(float));
  h_result = (float *)malloc(DSIZE*sizeof(float));

    
  // hipMalloc(&d_max_rand_int, sizeof(unsigned));
  // h_max_rand_int = (unsigned *)malloc(sizeof(unsigned));
  // hipMalloc(&d_min_rand_int, sizeof(unsigned));
  // h_min_rand_int = (unsigned *)malloc(sizeof(unsigned));
  hipMemset(d_result, 0, DSIZE*sizeof(float));
  setup_kernel<<<DSIZE,nTPB>>>(d_state,time(NULL));

  randfloat<<<DSIZE,nTPB>>>(d_state, d_result);

  hipMemcpy(h_result, d_result,DSIZE * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < DSIZE;)
  {
    printf("%.6f %.6f %.6f %.6f %.6f %.6f %.6f %.6f %.6f \n" ,
     h_result[i++], h_result[i++], h_result[i++],
     h_result[i++], h_result[i++], h_result[i++],
     h_result[i++], h_result[i++], h_result[i++]);
  }



/* 
  hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));
  unsigned *d_result, *h_result;
  unsigned *d_max_rand_int, *h_max_rand_int, *d_min_rand_int, *h_min_rand_int;
  hipMalloc(&d_result, (MAX-MIN+1) * sizeof(unsigned));
  h_result = (unsigned *)malloc((MAX-MIN+1)*sizeof(unsigned));
  hipMalloc(&d_max_rand_int, sizeof(unsigned));
  h_max_rand_int = (unsigned *)malloc(sizeof(unsigned));
  hipMalloc(&d_min_rand_int, sizeof(unsigned));
  h_min_rand_int = (unsigned *)malloc(sizeof(unsigned));
  hipMemset(d_result, 0, (MAX-MIN+1)*sizeof(unsigned));
  setup_kernel<<<1,1>>>(d_state,time(NULL));

  *h_max_rand_int = MAX;
  *h_min_rand_int = MIN;
  hipMemcpy(d_max_rand_int, h_max_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);
  hipMemcpy(d_min_rand_int, h_min_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);
  generate_kernel<<<1,1>>>(d_state, ITER, d_max_rand_int, d_min_rand_int, d_result);
  hipMemcpy(h_result, d_result, (MAX-MIN+1) * sizeof(unsigned), hipMemcpyDeviceToHost);
  printf("Bin:    Count: \n");
  for (int i = MIN; i <= MAX; i++)
    printf("%d    %d\n", i, h_result[i-MIN]);

  return 0;*/
}


// $ nvcc -arch=sm_20 -o t527 t527.cu -lcurand
// $ cuda-memcheck ./t527
// ========= CUDA-MEMCHECK
// Bin:    Count:
// 2    1665496
// 3    1668130
// 4    1667644
// 5    1667435
// 6    1665026
// 7    1666269
// ========= ERROR SUMMARY: 0 errors
// $