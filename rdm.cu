#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
#define MIN 1
#define MAX 10000
#define ITER 10000000
#define msg(format, ...) do { fprintf(stderr, format, ##__VA_ARGS__); } while (0)


__global__ void setup_kernel(hiprandState *state){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ void generate_kernel(hiprandState *my_curandstate, const unsigned int n,
                     const unsigned *max_rand_int, const unsigned *min_rand_int,  unsigned int *result){

  int idx = threadIdx.x + blockDim.x*blockIdx.x;

  int count = 0;
  while (count < n){
    float myrandf = hiprand_uniform(my_curandstate+idx);
    msg(myrandf);
    myrandf *= (max_rand_int[idx] - min_rand_int[idx]+0.999999);
    myrandf += min_rand_int[idx];
    int myrand = (int)truncf(myrandf);

    assert(myrand <= max_rand_int[idx]);
    assert(myrand >= min_rand_int[idx]);
    result[myrand-min_rand_int[idx]]++;
    count++;}
}

int main(){

  hiprandState *d_state;
  hipMalloc(&d_state, sizeof(hiprandState));
  
  unsigned *d_result, *h_result;//declare result variables device/host
  unsigned *d_max_rand_int, *h_max_rand_int, *d_min_rand_int, *h_min_rand_int;//declare device/host value variables 


  hipMalloc(&d_result, (MAX-MIN+1) * sizeof(unsigned)); //allocate memory to device Result

  h_result = (unsigned *)malloc((MAX-MIN+1)*sizeof(unsigned)); 
  
  hipMalloc(&d_max_rand_int, sizeof(unsigned));
  h_max_rand_int = (unsigned *)malloc(sizeof(unsigned));
  hipMalloc(&d_min_rand_int, sizeof(unsigned));  
  h_min_rand_int = (unsigned *)malloc(sizeof(unsigned));
  hipMemset(d_result, 0, (MAX-MIN+1)*sizeof(unsigned));
  setup_kernel<<<1,>>>(d_state);

  *h_max_rand_int = MAX;
  *h_min_rand_int = MIN;
  hipMemcpy(d_max_rand_int, h_max_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);//copy max to device
  hipMemcpy(d_min_rand_int, h_min_rand_int, sizeof(unsigned), hipMemcpyHostToDevice);//copy min to device

  generate_kernel<<<1,1>>>(d_state, ITER, d_max_rand_int, d_min_rand_int, d_result);
  
  hipMemcpy(h_result, d_result, (MAX-MIN+1) * sizeof(unsigned), hipMemcpyDeviceToHost);
  printf("Bin:    Count: \n");
  for (int i = MIN; i <= MAX; i++)
    printf("%d    %d\n", i, h_result[i-MIN]);

  return 0;
}


// $ nvcc -arch=sm_20 -o t527 t527.cu -lcurand
// $ cuda-memcheck ./t527
// ========= CUDA-MEMCHECK
// Bin:    Count:
// 2    1665496
// 3    1668130
// 4    1667644
// 5    1667435
// 6    1665026
// 7    1666269
// ========= ERROR SUMMARY: 0 errors
// $
